#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <math.h>
#include <cmath>
#include <chrono>
#include "common.h"
#include <iostream>
#include <vector>
#include <string>
#include <filesystem>
#define MAX_MASK_DIM 25

#define CHANNELS 3
#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

__constant__ float mask_c[MAX_MASK_DIM * MAX_MASK_DIM];

__global__ void batch_convolution_kernel(unsigned char *red, unsigned char *green, unsigned char *blue, unsigned char *gray,
                                         unsigned int *widths, unsigned int *heights, unsigned int *mem_offset, unsigned int mask_dimension)
{
    // Thread and block indices
    int image_idx = blockIdx.z;                        // Image index in the batch
    int row = blockIdx.y * BLOCK_SIZE_Y + threadIdx.y; // Row index
    int col = blockIdx.x * BLOCK_SIZE_X + threadIdx.x; // Column index
    int channel = threadIdx.z;                         // Channel index (0: red, 1: green, 2: blue)

    // Image dimensions and memory offset
    unsigned int width = widths[image_idx];
    unsigned int height = heights[image_idx];
    int image_offset = mem_offset[image_idx];

    unsigned int mask_radius = mask_dimension / 2;

    // Shared memory for intermediate results
    __shared__ float channel_sums[BLOCK_SIZE_Y][BLOCK_SIZE_X][CHANNELS];

    // Check if the current pixel is within bounds
    if (row < height && col < width)
    {
        float sum = 0.0f;

        // Apply the convolution mask
        for (int mask_row = 0; mask_row < mask_dimension; ++mask_row)
        {
            for (int mask_col = 0; mask_col < mask_dimension; ++mask_col)
            {
                int i_row = row - mask_radius + mask_row;
                int i_col = col - mask_radius + mask_col;

                if (i_row >= 0 && i_row < height && i_col >= 0 && i_col < width)
                {
                    if (channel == 0)
                    { // Red channel
                        sum += red[image_offset + i_row * width + i_col] * mask_c[mask_row * mask_dimension + mask_col];
                    }
                    else if (channel == 1)
                    { // Green channel
                        sum += green[image_offset + i_row * width + i_col] * mask_c[mask_row * mask_dimension + mask_col];
                    }
                    else if (channel == 2)
                    { // Blue channel
                        sum += blue[image_offset + i_row * width + i_col] * mask_c[mask_row * mask_dimension + mask_col];
                    }
                }
            }
        }

        // Clamp the result to [0, 255]
        channel_sums[threadIdx.y][threadIdx.x][channel] = fminf(fmaxf(sum, 0), 255);
        __syncthreads(); // Ensure all threads have written their results

        // Compute grayscale value (only thread with channel = 0 computes it)
        if (channel == 0)
        {
            float r = channel_sums[threadIdx.y][threadIdx.x][0];
            float g = channel_sums[threadIdx.y][threadIdx.x][1];
            float b = channel_sums[threadIdx.y][threadIdx.x][2];
            gray[image_offset + row * width + col] = static_cast<unsigned char>(0.299f * r + 0.587f * g + 0.114f * b);
        }
    }
}
void rgb2gray_gpu(float *mask, unsigned char *red, unsigned char *green, unsigned char *blue, unsigned char *gray,
                  unsigned int *widths, unsigned int *heights, unsigned int *mem_offset, unsigned int batch_size, unsigned int mask_dimension)
{
    using namespace std::chrono;

    unsigned char *red_d, *green_d, *blue_d, *gray_d;
    unsigned int *widths_d, *heights_d, *mem_offset_d;

    // Allocate memory on the device
    auto start = high_resolution_clock::now();
    hipMalloc((void **)&red_d, mem_offset[batch_size] * sizeof(unsigned char));
    hipMalloc((void **)&green_d, mem_offset[batch_size] * sizeof(unsigned char));
    hipMalloc((void **)&blue_d, mem_offset[batch_size] * sizeof(unsigned char));
    hipMalloc((void **)&gray_d, mem_offset[batch_size] * sizeof(unsigned char));
    hipMalloc((void **)&widths_d, batch_size * sizeof(unsigned int));
    hipMalloc((void **)&heights_d, batch_size * sizeof(unsigned int));
    hipMalloc((void **)&mem_offset_d, (batch_size + 1) * sizeof(unsigned int));
    hipDeviceSynchronize();
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<milliseconds>(stop - start).count();
    printf("Device Memory Allocation: %lld ms\n", duration);

    // Copy data to the device
    start = high_resolution_clock::now();
    hipMemcpy(red_d, red, mem_offset[batch_size] * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(green_d, green, mem_offset[batch_size] * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(blue_d, blue, mem_offset[batch_size] * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(widths_d, widths, batch_size * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(heights_d, heights, batch_size * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(mem_offset_d, mem_offset, (batch_size + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(mask_c), mask, mask_dimension * mask_dimension * sizeof(float));
    hipDeviceSynchronize();
    stop = high_resolution_clock::now();
    duration = duration_cast<milliseconds>(stop - start).count();
    printf("Data Transfer to Device: %lld ms\n", duration);

    // Determine the maximum image dimensions in the batch
    unsigned int max_width = 0, max_height = 0;
    for (size_t i = 0; i < batch_size; ++i)
    {
        if (widths[i] > max_width)
            max_width = widths[i];
        if (heights[i] > max_height)
            max_height = heights[i];
    }

    // Launch the kernel
    start = high_resolution_clock::now();
    dim3 numThreadsPerBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, CHANNELS);
    dim3 numBlocks((max_width + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X,
                   (max_height + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y,
                   batch_size);
    batch_convolution_kernel<<<numBlocks, numThreadsPerBlock>>>(red_d, green_d, blue_d, gray_d, widths_d, heights_d, mem_offset_d, mask_dimension);
    hipDeviceSynchronize();
    stop = high_resolution_clock::now();
    duration = duration_cast<milliseconds>(stop - start).count();
    printf("Kernel Execution: %lld ms\n", duration);

    // Check for CUDA errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
    else
    {
        printf("No CUDA Errors\n");
    }

    // Copy results back to the host
    start = high_resolution_clock::now();
    hipMemcpy(gray, gray_d, mem_offset[batch_size] * sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    stop = high_resolution_clock::now();
    duration = duration_cast<milliseconds>(stop - start).count();
    printf("Data Transfer to Host: %lld ms\n", duration);

    // Free device memory
    // start = high_resolution_clock::now();
    hipFree(red_d);
    hipFree(green_d);
    hipFree(blue_d);
    hipFree(gray_d);
    hipFree(widths_d);
    hipFree(heights_d);
    hipFree(mem_offset_d);
    hipDeviceSynchronize();
    // stop = high_resolution_clock::now();
    // duration = duration_cast<milliseconds>(stop - start).count();
    printf("Device Memory Deallocation: %lld ms\n", duration);
}

#define STB_IMAGE_IMPLEMENTATION
#include "dependencies/stb/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "dependencies/stb/stb_image_write.h"

namespace fs = std::filesystem;

// Count the number of images in the input folder
int count_images(const std::string &input_folder)
{
    int count = 0;
    for (const auto &entry : fs::directory_iterator(input_folder))
    {
        if (entry.is_regular_file())
        {
            count++;
        }
    }
    return count;
}

// Load a batch of images from the input folder and store their filenames
std::vector<std::pair<std::string, unsigned char *>> load_batch(const std::string &input_folder, int batch_size, int start_index, int total_images, int *width, int *height)
{
    std::vector<std::pair<std::string, unsigned char *>> batch;
    int count = 0;

    for (const auto &entry : fs::directory_iterator(input_folder))
    {
        if (count >= start_index && count < start_index + batch_size && count < total_images && entry.is_regular_file())
        {
            std::string file_path = entry.path().string();
            std::string filename = entry.path().filename().string(); // Extract the filename

            // Load image using STB
            int w, h, channels;
            unsigned char *image_data = stbi_load(file_path.c_str(), &w, &h, &channels, 0);
            if (image_data)
            {
                if (channels == 3 || channels == 4)
                { // Accept both RGB and RGBA images
                    unsigned char *rgb_data = image_data;
                    if (channels == 4)
                    {
                        // Convert RGBA to RGB by stripping the alpha channel
                        rgb_data = (unsigned char *)malloc(w * h * 3 * sizeof(unsigned char));
                        if (!rgb_data)
                        {
                            fprintf(stderr, "Error: Memory allocation failed for RGB conversion\n");
                            stbi_image_free(image_data);
                            continue;
                        }
                        for (int i = 0; i < w * h; ++i)
                        {
                            rgb_data[i * 3] = image_data[i * 4];         // Red channel
                            rgb_data[i * 3 + 1] = image_data[i * 4 + 1]; // Green channel
                            rgb_data[i * 3 + 2] = image_data[i * 4 + 2]; // Blue channel
                        }
                        stbi_image_free(image_data); // Free the original RGBA data
                    }
                    batch.emplace_back(filename, rgb_data);
                    width[count - start_index] = w;
                    height[count - start_index] = h;
                }
                else
                {
                    fprintf(stderr, "Skipping image %s: Expected 3 or 4 channels (RGB/RGBA), got %d channels\n", file_path.c_str(), channels);
                    stbi_image_free(image_data); // Free the image data if it doesn't match
                }
            }
            else
            {
                fprintf(stderr, "Failed to load image: %s\n", file_path.c_str());
            }
        }
        count++;
        if (batch.size() >= batch_size || count >= total_images)
        {
            break;
        }
    }

    return batch;
}

// Function to save a batch of grayscale images
void save_batch(const unsigned char *gray, unsigned int *widths, unsigned int *heights, unsigned int *mem_offset, const std::string &output_folder, const std::vector<std::string> &filenames, const char *prefix_id)
{
    for (size_t i = 0; i < filenames.size(); ++i)
    {
        std::string output_path = output_folder + "/" + prefix_id + filenames[i];
        stbi_write_png(output_path.c_str(), widths[i], heights[i], 1, gray + mem_offset[i], widths[i]);
    }
}

int main(int argc, char *argv[])
{
    printf("----- Kernel 1 (Different Size Images) -----\n");
    // Process command-line arguments
    if (argc < 5)
    {
        fprintf(stderr, "Usage: %s <input_folder> <output_folder> <batch_size> <mask_file_path>\n", argv[0]);
        return 1;
    }

    std::string input_folder = argv[1];
    std::string output_folder = argv[2];
    unsigned int batch_size = std::stoi(argv[3]);
    const char *mask_file = argv[4];

    // Count the total number of images in the input folder
    int total_images = count_images(input_folder);
    if (total_images == 0)
    {
        fprintf(stderr, "No images found in the input folder.\n");
        return 1;
    }

    printf("Found %d images in the input folder.\n", total_images);
    printf("--------------------------------------------------\n");

    // Calculate the number of batches
    int num_batches = std::ceil((float)total_images / batch_size);

    int mask_dimension;
    float **mask = read_mask(mask_file, &mask_dimension);
    float *flattened_mask = flatten_mask(mask, mask_dimension);

    const char *prefix_id = ""; // Prefix for the output filenames

    // Batches are processed sequentially
    for (int batch_idx = 0; batch_idx < num_batches; ++batch_idx)
    {
        printf("*********** Processing batch %d ************\n", batch_idx + 1);

        int start_index = batch_idx * batch_size;

        int batch_size_actual = (start_index + batch_size > total_images) ? (total_images - start_index) : batch_size;

        // Load the current batch of images and filenames
        int *widths = (int *)malloc(batch_size_actual * sizeof(int));
        int *heights = (int *)malloc(batch_size_actual * sizeof(int));
        std::vector<std::pair<std::string, unsigned char *>> batch = load_batch(input_folder, batch_size_actual, start_index, total_images, widths, heights);
        if (batch.empty())
        {
            fprintf(stderr, "Failed to load batch %d\n", batch_idx);
            continue;
        }

        // Calculate memory offsets
        unsigned int *mem_offset = (unsigned int *)malloc((batch_size_actual + 1) * sizeof(unsigned int));
        mem_offset[0] = 0;
        for (size_t i = 1; i <= batch_size_actual; ++i)
        {
            mem_offset[i] = mem_offset[i - 1] + widths[i - 1] * heights[i - 1];
        }

        // Collect filenames
        std::vector<std::string> filenames;
        for (const auto &pair : batch)
        {
            filenames.push_back(pair.first); // Save the filenames
        }

        // Allocate memory for the batch of grayscale images
        unsigned char *gray = (unsigned char *)malloc(mem_offset[batch_size_actual] * sizeof(unsigned char));
        if (!gray)
        {
            fprintf(stderr, "Error: Memory allocation failed for grayscale images\n");
            return 1;
        }

        // Allocate memory for the batch of RGB images
        unsigned char *red = (unsigned char *)malloc(mem_offset[batch_size_actual] * sizeof(unsigned char));
        unsigned char *green = (unsigned char *)malloc(mem_offset[batch_size_actual] * sizeof(unsigned char));
        unsigned char *blue = (unsigned char *)malloc(mem_offset[batch_size_actual] * sizeof(unsigned char));
        if (!red || !green || !blue)
        {
            fprintf(stderr, "Error: Memory allocation failed for RGB images\n");
            free(gray);
            return 1;
        }

        // Copy RGB data from the batch to the flat arrays
        for (size_t i = 0; i < batch.size(); ++i)
        {
            unsigned char *image_data = batch[i].second;
            for (int j = 0; j < widths[i] * heights[i]; ++j)
            {
                red[mem_offset[i] + j] = image_data[j * 3];       // Red channel
                green[mem_offset[i] + j] = image_data[j * 3 + 1]; // Green channel
                blue[mem_offset[i] + j] = image_data[j * 3 + 2];  // Blue channel
            }
        }

        // Call the GPU function (process the entire batch in one call)
        rgb2gray_gpu(flattened_mask, red, green, blue, gray, (unsigned int *)widths, (unsigned int *)heights, mem_offset, batch_size_actual, mask_dimension);

        // Save the batch of grayscale images with original filenames
        save_batch(gray, (unsigned int *)widths, (unsigned int *)heights, mem_offset, output_folder, filenames, prefix_id);

        // Free memory
        free(red);
        free(green);
        free(blue);
        free(gray);
        for (const auto &pair : batch)
        {
            if (pair.second)
            {
                free(pair.second); // Free the RGB data (may have been allocated for RGBA conversion)
            }
        }

        printf("Batch %d processed and saved.\n", batch_idx + 1);
        printf("--------------------------------------------------\n");
    }

    // Free the mask memory
    for (int i = 0; i < mask_dimension; ++i)
    {
        free(mask[i]);
    }
    free(mask);
    free(flattened_mask);

    printf("Grayscale conversion completed. Results saved to %s.\n", output_folder.c_str());
    return 0;
}

/*
cd E:\Spring25\Selected_topics\Lab5
nvcc -o k1 kernel1_m.cu -std=c++17
kernel2 input_images output_images/kernel2_cu 2 masks/mask.txt
*/